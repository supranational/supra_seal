#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <iostream>
#include <thread>
#include <vector>
#include <mutex>
#include <algorithm>

#define SPPARK_DONT_INSTANTIATE_TEMPLATES

//#define G2_MSM_ON_GPU

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381-fp2.hpp>
#else
# error "only FEATURE_BLS12_381 is supported"
#endif

#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;

typedef jacobian_t<fp2_t> point_fp2_t;
typedef xyzz_t<fp2_t> bucket_fp2_t;
typedef bucket_fp2_t::affine_t affine_fp2_t;

typedef fr_t scalar_t;

#include <msm/pippenger.cuh>
#include <msm/pippenger.hpp>

#ifdef __CUDA_ARCH__
typedef uint8_t byte;
#endif

template<class Scalar>
struct Assignment {
    // Density of queries
    const uint64_t* a_aux_density;
    size_t a_aux_bit_len;
    size_t a_aux_popcount;

    const uint64_t* b_inp_density;
    size_t b_inp_bit_len;
    size_t b_inp_popcount;

    const uint64_t* b_aux_density;
    size_t b_aux_bit_len;
    size_t b_aux_popcount;

    // Evaluations of A, B, C polynomials
    const Scalar* a;
    const Scalar* b;
    const Scalar* c;
    size_t abc_size;

    // Assignments of variables
    const Scalar* inp_assignment_data;
    size_t inp_assignment_size;

    const Scalar* aux_assignment_data;
    size_t aux_assignment_size;
};

#include "groth16_ntt_h.cu"
#include "groth16_split_msm.cu"

template<class point_t, class affine_t>
static void mult(point_t& ret, const affine_t point, const scalar_t& fr,
                 size_t top = scalar_t::nbits)
{
#ifndef __CUDA_ARCH__
    scalar_t::pow_t scalar;
    fr.to_scalar(scalar);

    mult(ret, point, scalar, top);
#endif
}

static thread_pool_t groth16_pool;

struct msm_results {
    std::vector<point_t> h;
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    msm_results(size_t num_circuits) : h(num_circuits),
                                       l(num_circuits),
                                       a(num_circuits),
                                       b_g1(num_circuits),
                                       b_g2(num_circuits) {}
};

struct groth16_proof {
    point_t::affine_t a;
    point_fp2_t::affine_t b;
    point_t::affine_t c;
};

#include "groth16_srs.cuh"

#if defined(_MSC_VER) && !defined(__clang__) && !defined(__builtin_popcountll)
#define __builtin_popcountll(x) __popcnt64(x)
#endif

extern "C"
RustError::by_value generate_groth16_proofs_c(const Assignment<fr_t> provers[],
                                              size_t num_circuits,
                                              const fr_t r_s[], const fr_t s_s[],
                                              groth16_proof proofs[], SRS& srs)
{
    // Mutex to serialize execution of this subroutine
    static std::mutex mtx;
    std::lock_guard<std::mutex> lock(mtx);

    if (!ngpus()) {
        return RustError{ENODEV, "No CUDA devices available"};
    }

    const verifying_key* vk = &srs.get_vk();

    auto points_h = srs.get_h_slice();
    auto points_l = srs.get_l_slice();
    auto points_a = srs.get_a_slice();
    auto points_b_g1 = srs.get_b_g1_slice();
    auto points_b_g2 = srs.get_b_g2_slice();

    for (size_t c = 0; c < num_circuits; c++) {
        auto& p = provers[c];

        assert(points_l.size() == p.aux_assignment_size);
        assert(points_a.size() == p.inp_assignment_size + p.a_aux_popcount);
        assert(points_b_g1.size() == p.b_inp_popcount + p.b_aux_popcount);
        assert(p.a_aux_bit_len == p.aux_assignment_size);
        assert(p.b_aux_bit_len == p.aux_assignment_size);
        assert(p.b_inp_bit_len == p.inp_assignment_size);
    }

    bool l_skip_batch_add = false, a_skip_batch_add = false,
         b_skip_batch_add = false;

    split_vectors split_vectors_l{num_circuits, points_l.size()};
    split_vectors split_vectors_a{num_circuits, points_a.size()};
    split_vectors split_vectors_b{num_circuits, points_b_g1.size()};

    std::vector<affine_t> tail_msm_l_bases,
                          tail_msm_a_bases,
                          tail_msm_b_g1_bases;
    std::vector<affine_fp2_t> tail_msm_b_g2_bases;

    msm_results results{num_circuits};

    semaphore_t barrier;
    std::atomic<bool> caught_exception{false};
    size_t n_gpus = std::min(ngpus(), num_circuits);

    std::thread prep_msm_thread([&, num_circuits]
    {
        // pre-processing step
        // mark inp and significant scalars in aux assignments
        groth16_pool.par_map(num_circuits, [&](size_t c) {
            auto& prover = provers[c];
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];

            size_t a_bits_cursor = 0, b_bits_cursor = 0;
            uint64_t a_bits = 0, b_bits = 0;
            uint32_t a_bit_off = 0, b_bit_off = 0;

            size_t inp_size = prover.inp_assignment_size;

            for (size_t i = 0; i < inp_size; i += CHUNK_BITS) {
                uint64_t b_map = prover.b_inp_density[i / CHUNK_BITS];
                uint64_t map_mask = 1;
                size_t chunk_bits = std::min(CHUNK_BITS, inp_size - i);

                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    a_bits |= map_mask;

                    if (b_map & map_mask) {
                        b_bits |= (uint64_t)1 << b_bit_off;
                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                        }
                    }
                }

                a_bit_vector[i / CHUNK_BITS] = a_bits;
                if (chunk_bits == CHUNK_BITS)
                    a_bits = 0;
            }

            a_bits_cursor = inp_size / CHUNK_BITS;
            a_bit_off = inp_size % CHUNK_BITS;

            auto* aux_assignment = prover.aux_assignment_data;
            size_t aux_size = prover.aux_assignment_size;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = prover.a_aux_density[i / CHUNK_BITS];
                uint64_t b_map = prover.b_aux_density[i / CHUNK_BITS];
                uint64_t l_bits = 0;
                uint64_t map_mask = 1;
                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);

                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];

                    bool is_one = scalar.is_one();
                    bool is_zero = scalar.is_zero();

                    if (!is_zero && !is_one)
                        l_bits |= map_mask;

                    if (a_map & map_mask) {
                        if (!is_zero && !is_one) {
                            a_bits |= ((uint64_t)1 << a_bit_off);
                        }

                        if (++a_bit_off == CHUNK_BITS) {
                            a_bit_off = 0;
                            a_bit_vector[a_bits_cursor++] = a_bits;
                            a_bits = 0;
                        }
                    }

                    if (b_map & map_mask) {
                        if (!is_zero && !is_one) {
                            b_bits |= ((uint64_t)1 << b_bit_off);
                        }

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = b_bits;
                            b_bits = 0;
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;
        });

        if (caught_exception)
            return;

        // merge all the masks from aux_assignments and count set bits
        std::vector<mask_t> tail_msm_l_mask(split_vectors_l.bit_vector_size);
        std::vector<mask_t> tail_msm_a_mask(split_vectors_a.bit_vector_size);
        std::vector<mask_t> tail_msm_b_mask(split_vectors_b.bit_vector_size);

        size_t l_popcount = 0, a_popcount = 0, b_popcount = 0;

        for (size_t i = 0; i < tail_msm_l_mask.size(); i++) {
            uint64_t mask = split_vectors_l.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_l.bit_vector[c][i];
            tail_msm_l_mask[i] = mask;
            l_popcount += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_a_mask.size(); i++) {
            uint64_t mask = split_vectors_a.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_a.bit_vector[c][i];
            tail_msm_a_mask[i] = mask;
            a_popcount += __builtin_popcountll(mask);
        }

        for (size_t i = 0; i < tail_msm_b_mask.size(); i++) {
            uint64_t mask = split_vectors_b.bit_vector[0][i];
            for (size_t c = 1; c < num_circuits; c++)
                mask |= split_vectors_b.bit_vector[c][i];
            tail_msm_b_mask[i] = mask;
            b_popcount += __builtin_popcountll(mask);
        }

        if (caught_exception)
            return;

        if (!(l_skip_batch_add = (l_popcount >= points_l.size() / 2))) {
            split_vectors_l.tail_msms_resize(l_popcount);
            tail_msm_l_bases.resize(l_popcount);
        }

        if (a_skip_batch_add = (a_popcount >= points_a.size() / 2)) {
            split_vectors_a.tail_msms_resize(points_a.size());
        }
        else {
            split_vectors_a.tail_msms_resize(a_popcount);
            tail_msm_a_bases.resize(a_popcount);
        }

        if (b_skip_batch_add = (b_popcount >= points_b_g1.size() / 2)) {
            split_vectors_b.tail_msms_resize(points_b_g1.size());
        }
        else {
            split_vectors_b.tail_msms_resize(b_popcount);
            tail_msm_b_g1_bases.resize(b_popcount);
            tail_msm_b_g2_bases.resize(b_popcount);
        }

        // populate bitmaps for batch additions, bases and scalars for tail msms
        groth16_pool.par_map(num_circuits, [&](size_t c) {
            auto& prover = provers[c];
            auto& l_bit_vector = split_vectors_l.bit_vector[c];
            auto& a_bit_vector = split_vectors_a.bit_vector[c];
            auto& b_bit_vector = split_vectors_b.bit_vector[c];
            auto& tail_msm_l_scalars = split_vectors_l.tail_msm_scalars[c];
            auto& tail_msm_a_scalars = split_vectors_a.tail_msm_scalars[c];
            auto& tail_msm_b_scalars = split_vectors_b.tail_msm_scalars[c];

            size_t a_cursor = 0, b_cursor = 0;

            uint32_t a_bit_off = 0, b_bit_off = 0;
            size_t a_bits_cursor = 0, b_bits_cursor = 0;

            auto* inp_assignment = prover.inp_assignment_data;
            size_t inp_size = prover.inp_assignment_size;

            for (size_t i = 0; i < inp_size; i += CHUNK_BITS) {
                uint64_t b_map = prover.b_inp_density[i / CHUNK_BITS];
                size_t chunk_bits = std::min(CHUNK_BITS, inp_size - i);

                for (size_t j = 0; j < chunk_bits; j++, b_map >>= 1) {
                    const fr_t& scalar = inp_assignment[i + j];

                    if (b_map & 1) {
                        if (c == 0 && !b_skip_batch_add) {
                            tail_msm_b_g1_bases[b_cursor] = points_b_g1[b_cursor];
                            tail_msm_b_g2_bases[b_cursor] = points_b_g2[b_cursor];
                        }
                        tail_msm_b_scalars[b_cursor] = scalar;
                        b_cursor++;

                        if (++b_bit_off == CHUNK_BITS) {
                            b_bit_off = 0;
                            b_bit_vector[b_bits_cursor++] = 0;
                        }
                    }

                    if (c == 0 && !a_skip_batch_add)
                        tail_msm_a_bases[a_cursor] = points_a[a_cursor];
                    tail_msm_a_scalars[a_cursor] = scalar;
                    a_cursor++;
                }

                a_bit_vector[i / CHUNK_BITS] = 0;
            }

            assert(b_cursor == prover.b_inp_popcount);

            a_bits_cursor = inp_size / CHUNK_BITS;
            a_bit_off = inp_size % CHUNK_BITS;

            uint64_t a_mask = tail_msm_a_mask[a_bits_cursor], a_bits = 0;
            uint64_t b_mask = tail_msm_b_mask[b_bits_cursor], b_bits = 0;

            size_t points_a_cursor = a_cursor,
                   points_b_cursor = b_cursor,
                   l_cursor = 0;

            auto* aux_assignment = prover.aux_assignment_data;
            size_t aux_size = prover.aux_assignment_size;

            for (size_t i = 0; i < aux_size; i += CHUNK_BITS) {
                uint64_t a_map = prover.a_aux_density[i / CHUNK_BITS];
                uint64_t b_map = prover.b_aux_density[i / CHUNK_BITS];
                uint64_t l_map = tail_msm_l_mask[i / CHUNK_BITS], l_bits = 0;
                uint64_t map_mask = 1;

                size_t chunk_bits = std::min(CHUNK_BITS, aux_size - i);
                for (size_t j = 0; j < chunk_bits; j++, map_mask <<= 1) {
                    const fr_t& scalar = aux_assignment[i + j];
                    bool is_one = scalar.is_one();

                    if (!l_skip_batch_add) {
                        if (is_one)
                            l_bits |= map_mask;

                        if (l_map & map_mask) {
                            if (c == 0)
                                tail_msm_l_bases[l_cursor] = points_l[i+j];
                            tail_msm_l_scalars[l_cursor] = czero(scalar, is_one);
                            l_cursor++;
                        }
                    }

                    if (!a_skip_batch_add) {
                        if (a_map & map_mask) {
                            uint64_t mask = (uint64_t)1 << a_bit_off;

                            if (a_mask & mask) {
                                if (c == 0)
                                    tail_msm_a_bases[a_cursor] = points_a[points_a_cursor];
                                tail_msm_a_scalars[a_cursor] = czero(scalar, is_one);
                                a_cursor++;
                            }

                            points_a_cursor++;

                            if (is_one)
                                a_bits |= mask;

                            if (++a_bit_off == CHUNK_BITS) {
                                a_bit_off = 0;
                                a_bit_vector[a_bits_cursor++] = a_bits;
                                a_bits = 0;
                                a_mask = tail_msm_a_mask[a_bits_cursor];
                            }
                        }
                    }
                    else {
                        if (a_map & map_mask) {
                            tail_msm_a_scalars[a_cursor] = scalar;
                            a_cursor++;
                        }
                    }

                    if (!b_skip_batch_add) {
                        if (b_map & map_mask) {
                            uint64_t mask = (uint64_t)1 << b_bit_off;

                            if (b_mask & mask) {
                                if (c == 0) {
                                    tail_msm_b_g1_bases[b_cursor] =
                                        points_b_g1[points_b_cursor];
                                    tail_msm_b_g2_bases[b_cursor] =
                                        points_b_g2[points_b_cursor];
                                }
                                tail_msm_b_scalars[b_cursor] = czero(scalar,
                                                                     is_one);
                                b_cursor++;
                            }

                            points_b_cursor++;

                            if (is_one)
                                b_bits |= mask;

                            if (++b_bit_off == CHUNK_BITS) {
                                b_bit_off = 0;
                                b_bit_vector[b_bits_cursor++] = b_bits;
                                b_bits = 0;
                                b_mask = tail_msm_b_mask[b_bits_cursor];
                            }
                        }
                    }
                    else {
                        if (b_map & map_mask) {
                            tail_msm_b_scalars[b_cursor] = scalar;
                            b_cursor++;
                        }
                    }
                }

                l_bit_vector[i / CHUNK_BITS] = l_bits;
            }

            if (a_bit_off)
                a_bit_vector[a_bits_cursor] = a_bits;

            if (b_bit_off)
                b_bit_vector[b_bits_cursor] = b_bits;

            if (!l_skip_batch_add)
                assert(l_cursor == l_popcount);

            if (!a_skip_batch_add) {
                assert(points_a_cursor == points_a.size());
                assert(a_cursor == a_popcount);
            }
            else {
                assert(a_cursor == points_a.size());
            }

            if (!b_skip_batch_add) {
                assert(points_b_cursor == points_b_g1.size());
                assert(b_cursor == b_popcount);
            }
            else {
                assert(b_cursor == points_b_g1.size());
            }

        });
        // end of pre-processing step

        for (size_t i = 0; i < n_gpus; i++)
            barrier.notify();

        if (caught_exception)
            return;

#ifndef G2_MSM_ON_GPU
            // tail MSM b_g2 - on CPU
            for (size_t c = 0; c < num_circuits; c++) {
# ifndef __CUDA_ARCH__
                mult_pippenger<bucket_fp2_t>(results.b_g2[c],
                    b_skip_batch_add ? points_b_g2.data() :
                                       tail_msm_b_g2_bases.data(),
                    split_vectors_b.tail_msm_scalars[c].size(),
                    split_vectors_b.tail_msm_scalars[c].data(),
                    true, &groth16_pool);
# endif

                if (caught_exception)
                    return;
            }
#endif
    });

    batch_add_results batch_add_res{num_circuits};
    std::vector<std::thread> per_gpu;
    RustError ret{hipSuccess};

    for (size_t tid = 0; tid < n_gpus; tid++) {
        per_gpu.emplace_back(std::thread([&, tid, n_gpus](size_t num_circuits)
        {
            const gpu_t& gpu = select_gpu(tid);

            size_t rem = num_circuits % n_gpus;
            num_circuits /= n_gpus;
            num_circuits += tid < rem;
            size_t circuit0 = tid * num_circuits;
            if (tid >= rem)
                circuit0 += rem;

            try {
                {
                    size_t d_a_sz = sizeof(fr_t) << (lg2(points_h.size() - 1) + 1);
                    gpu_ptr_t<fr_t> d_a{(scalar_t*)gpu.Dmalloc(d_a_sz)};

                    for (size_t c = circuit0; c < circuit0 + num_circuits; c++) {
#ifndef __CUDA_ARCH__
                        ntt_msm_h::execute_ntt_msm_h(gpu, d_a, provers[c],
                                                     points_h.data(), points_h.size(),
                                                     results.h[c]);
#endif
                        if (caught_exception)
                            return;
                    }
                }

                barrier.wait();

                if (caught_exception)
                    return;

                if (!l_skip_batch_add) {
                    // batch addition L - on GPU
                    execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                        points_l.data(), points_l.size(), split_vectors_l,
                        &batch_add_res.l[circuit0]);

                    if (caught_exception)
                        return;
                }

                if (!a_skip_batch_add) {
                    // batch addition a - on GPU
                    execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                        points_a.data(), points_a.size(), split_vectors_a,
                        &batch_add_res.a[circuit0]);

                    if (caught_exception)
                        return;
                }

                if (!b_skip_batch_add) {
                    // batch addition b_g1 - on GPU
                    execute_batch_addition<bucket_t>(gpu, circuit0, num_circuits,
                        points_b_g1.data(), points_b_g1.size(), split_vectors_b,
                        &batch_add_res.b_g1[circuit0]);

                    if (caught_exception)
                        return;

                    // batch addition b_g2 - on GPU
                    execute_batch_addition<bucket_fp2_t>(gpu, circuit0,
                        num_circuits, points_b_g2.data(), points_b_g2.size(),
                        split_vectors_b, &batch_add_res.b_g2[circuit0]);

                    if (caught_exception)
                        return;
                }

                {
                    msm_t<bucket_t, point_t, affine_t, scalar_t> msm{nullptr,
                        std::max({l_skip_batch_add ? points_l.size() :
                                  split_vectors_l.tail_msm_scalars[0].size(),
                                  split_vectors_a.tail_msm_scalars[0].size(),
                                  split_vectors_b.tail_msm_scalars[0].size()})};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM l - on GPU
                        if (!l_skip_batch_add)
                            msm.invoke(results.l[c], tail_msm_l_bases,
                                split_vectors_l.tail_msm_scalars[c], true);
                        else
                            msm.invoke(results.l[c], points_l.data(), points_l.size(),
                                provers[c].aux_assignment_data, true);

                        if (caught_exception)
                            return;

                        // tail MSM a - on GPU
                        if (!a_skip_batch_add)
                            msm.invoke(results.a[c], tail_msm_a_bases,
                                split_vectors_a.tail_msm_scalars[c], true);
                        else
                            msm.invoke(results.a[c], points_a.data(), points_a.size(),
                                split_vectors_a.tail_msm_scalars[c].data(), true);

                        if (caught_exception)
                            return;

                        // tail MSM b_g1 - on GPU
                        if (!b_skip_batch_add)
                            msm.invoke(results.b_g1[c], tail_msm_b_g1_bases,
                                split_vectors_b.tail_msm_scalars[c], true);
                        else
                            msm.invoke(results.b_g1[c], points_b_g1.data(),
                                points_b_g1.size(),
                                split_vectors_b.tail_msm_scalars[c].data(), true);

                        if (caught_exception)
                            return;
                    }
                }

#ifdef G2_MSM_ON_GPU
                {
                    msm_t<bucket_fp2_t, point_fp2_t, affine_fp2_t, scalar_t>
                        msm{nullptr, split_vectors_b.tail_msm_scalars[0].size()};

                    for (size_t c = circuit0; c < circuit0+num_circuits; c++) {
                        // tail MSM b_g2 - on GPU
                        if (!b_skip_batch_add)
                            msm.invoke(results.b_g2[c], tail_msm_b_g2_bases,
                                split_vectors_b.tail_msm_scalars[c], true);
                        else
                            msm.invoke(results.b_g2[c], points_b_g2.data(),
                                points_b_g2.size(),
                                split_vectors_b.tail_msm_scalars[c].data(), true);

                        if (caught_exception)
                            return;
                    }
                }
#endif
            } catch (const cuda_error& e) {
                bool already = caught_exception.exchange(true);
                if (!already) {
                    for (size_t i = 1; i < n_gpus; i++)
                        barrier.notify();
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
                    ret = RustError{e.code(), e.what()};
#else
                    ret = RustError{e.code()};
#endif
                }
                gpu.sync();
            }
        }, num_circuits));
    }

    prep_msm_thread.join();
    for (auto& tid : per_gpu)
        tid.join();

    if (caught_exception)
        return ret;

    for (size_t circuit = 0; circuit < num_circuits; circuit++) {
        if (!l_skip_batch_add)
            results.l[circuit].add(batch_add_res.l[circuit]);
        if (!a_skip_batch_add)
            results.a[circuit].add(batch_add_res.a[circuit]);
        if (!b_skip_batch_add) {
            results.b_g1[circuit].add(batch_add_res.b_g1[circuit]);
            results.b_g2[circuit].add(batch_add_res.b_g2[circuit]);
        }

        fr_t r = r_s[circuit], s = s_s[circuit];
        fr_t rs = r * s;
        // we want the scalars to be in Montomery form when passing them to
        // "mult" routine

        point_t g_a, g_c, a_answer, b1_answer, vk_delta_g1_rs, vk_alpha_g1_s,
                vk_beta_g1_r;
        point_fp2_t g_b;

        mult(vk_delta_g1_rs, vk->delta_g1, rs);
        mult(vk_alpha_g1_s, vk->alpha_g1, s);
        mult(vk_beta_g1_r, vk->beta_g1, r);

        mult(b1_answer, results.b_g1[circuit], r);

        // A
        mult(g_a, vk->delta_g1, r);
        g_a.add(vk->alpha_g1);
        g_a.add(results.a[circuit]);

        // B
        mult(g_b, vk->delta_g2, s);
        g_b.add(vk->beta_g2);
        g_b.add(results.b_g2[circuit]);

        // C
        mult(g_c, results.a[circuit], s);
        g_c.add(b1_answer);
        g_c.add(vk_delta_g1_rs);
        g_c.add(vk_alpha_g1_s);
        g_c.add(vk_beta_g1_r);
        g_c.add(results.h[circuit]);
        g_c.add(results.l[circuit]);

        // to affine
        proofs[circuit].a = g_a;
        proofs[circuit].b = g_b;
        proofs[circuit].c = g_c;
    }

    return ret;
}
