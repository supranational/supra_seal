#include "hip/hip_runtime.h"
// Copyright Supranational LLC

#include <msm/batch_addition.cuh>

template __global__
void batch_addition<bucket_t>(bucket_t::mem_t ret_[],
                              const affine_t::mem_t points_[], uint32_t npoints,
                              const uint32_t bitmap[], bool accumulate,
                              uint32_t sid);

template __global__
void batch_addition<bucket_fp2_t>(bucket_fp2_t::mem_t ret_[],
                                  const affine_fp2_t::mem_t points_[],
                                  uint32_t npoints, const uint32_t bitmap[],
                                  bool accumulate, uint32_t sid);

struct batch_add_results {
    std::vector<point_t> l;
    std::vector<point_t> a;
    std::vector<point_t> b_g1;
    std::vector<point_fp2_t> b_g2;

    batch_add_results(size_t num_circuits) : l(num_circuits),
                                             a(num_circuits),
                                             b_g1(num_circuits),
                                             b_g2(num_circuits) { }
};

template<typename T> class uninit {
    T val;
public:
    uninit()            { } // don't zero std::vector<uninit<T>>
    uninit(T v)         { val = v; }
    operator T() const  { return val; }
};

using mask_t = uninit<uint64_t>;

const size_t CHUNK_BITS = sizeof(mask_t) * 8; // 64 bits

#define NUM_BATCHES 8
#define GPU_DIV (32*WARP_SZ)

class split_vectors {
public:
    std::vector<std::vector<mask_t>> bit_vector;
    std::vector<std::vector<fr_t>>   tail_msm_scalars;
    size_t batch_size, bit_vector_size;

    split_vectors(size_t num_circuits, size_t num_points)
        :   bit_vector{num_circuits},
            tail_msm_scalars{num_circuits}
    {
        batch_size = (num_points + GPU_DIV - 1) / GPU_DIV;
        batch_size = (batch_size + NUM_BATCHES - 1) / NUM_BATCHES;
        batch_size *= GPU_DIV;

        bit_vector_size = (num_points + CHUNK_BITS - 1) / CHUNK_BITS;

        for (size_t c = 0; c < num_circuits; c++) {
            bit_vector[c].resize(bit_vector_size);
        }
    }

    void tail_msms_resize(size_t num_sig_scalars) {
        size_t num_circuits = tail_msm_scalars.size();
        for (size_t c = 0; c < num_circuits; c++) {
            tail_msm_scalars[c].resize(num_sig_scalars);
        }
    }
};

template<class bucket_t,
         class point_t,
         class bucket_h = class bucket_t::mem_t,
         class affine_t = class bucket_t::affine_t,
         class affine_h = class bucket_t::affine_t::mem_t>
void execute_batch_addition(const gpu_t& gpu,
                            size_t circuit0, size_t num_circuits,
                            const affine_t points[], size_t npoints,
                            const split_vectors& split_vector,
                            point_t batch_add_res[])
{
    int sm_count = gpu.sm_count();

    uint32_t nbuckets = sm_count * BATCH_ADD_BLOCK_SIZE / WARP_SZ;

    uint32_t bit_vector_size = (split_vector.bit_vector_size + WARP_SZ - 1) & (0u - WARP_SZ);
    size_t batch_size = split_vector.batch_size;

    assert(batch_size == (uint32_t)batch_size);

    size_t d_points_size = batch_size * 2 * sizeof(affine_h);
    size_t d_buckets_size = num_circuits * nbuckets * sizeof(bucket_h);

    dev_ptr_t<uint8_t> d_temp{d_points_size + d_buckets_size +
                              num_circuits * bit_vector_size * sizeof(mask_t)};

    vec2d_t<affine_h> d_points{&d_temp[0], (uint32_t)batch_size};
    vec2d_t<bucket_h> d_buckets{&d_temp[d_points_size], nbuckets};
    vec2d_t<mask_t>   d_bit_vectors{&d_temp[d_points_size + d_buckets_size],
                                    bit_vector_size};

    uint32_t sid = 0;

    for (size_t c = 0; c < num_circuits; c++)
        gpu[sid].HtoD(d_bit_vectors[c],
                      split_vector.bit_vector[circuit0 + c].data(),
                      split_vector.bit_vector[circuit0 + c].size());

    for (uint32_t batch = 0; npoints > 0; batch++, sid ^= 1) {
        uint32_t amount = std::min(npoints, batch_size);
        size_t cursor = batch * batch_size;

        gpu[sid].HtoD(d_points[sid], &points[cursor], amount);

        for (size_t c = 0; c < num_circuits; c++)
            gpu[sid].launch_coop(batch_addition<bucket_t>,
                {sm_count, BATCH_ADD_BLOCK_SIZE},
                d_buckets[c], (const affine_h*)d_points[sid], amount,
                (const uint32_t*)&d_bit_vectors[c][cursor / CHUNK_BITS],
                batch > 0, sid);

        npoints -= amount;
    }
    sid ^= 1;

    vec2d_t<bucket_t> buckets{nbuckets, num_circuits};
    gpu[sid].DtoH(buckets[0], d_buckets[0], num_circuits * nbuckets);
    gpu[sid].sync();

    gpu.par_map(num_circuits, 1, [&, batch_add_res, nbuckets](size_t c) {
        batch_add_res[c] = sum_up(buckets[c], nbuckets);
    });
}
